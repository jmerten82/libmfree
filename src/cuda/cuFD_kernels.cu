#include "hip/hip_runtime.h"
/*** /cuda/cuda_cuFD_kernels.cu
     These are the CUDA kernels which are associated with
     the libmfree cuFD host drivers.

Julian Merten
INAF OA Bologna
Jul 2018
julian.merten@oabo.inaf.it
http://www.julianmerten.net
***/

#include <mfree/cuda/cuFD_kernels.h>

__global__ void cuFD_weights_set_zeros(int matrix_stride, int polynomial, int nn,  double *A)
{
  
  int offsetA = blockIdx.x*matrix_stride*matrix_stride + (nn+threadIdx.x)*matrix_stride + nn;
  
  for(int i = 0; i < polynomial; i++)
    {
      A[offsetA+i] = 0.;
    }
}

__global__ void cuFD_ga_weights_matrix_part(int* tree, double *all_coordinates, double *shapes,int matrix_stride, int pdeg, double* A)
{
int offsetA = blockIdx.x*matrix_stride*matrix_stride;

  //Getting all nearest neighbours in the shared memory
  __shared__ double coordinates[MAX_NN][2];
  int index = tree[blockIdx.x*blockDim.x+threadIdx.x];
  coordinates[threadIdx.x][0] = all_coordinates[index*2] - all_coordinates[blockIdx.x*2];
  coordinates[threadIdx.x][1] = all_coordinates[index*2+1] - all_coordinates[blockIdx.x*2+1];
  __syncthreads();

  row_vector_from_polynomial(coordinates[threadIdx.x][0],coordinates[threadIdx.x][1],matrix_stride, pdeg,&A[offsetA+threadIdx.x*matrix_stride+blockDim.x],&A[offsetA+blockDim.x*matrix_stride+threadIdx.x]);

  //Running over all neighbours in this point.
  for(int i = 0; i < blockDim.x; i++)
    {
      double x = coordinates[i][0] - coordinates[threadIdx.x][0];
      double y = coordinates[i][1] - coordinates[threadIdx.x][1];
      //Setting changing part of coefficient matrix
      A[offsetA+threadIdx.x*matrix_stride+i] = ga(x,y,shapes[blockIdx.x]);
    }
}

__global__ void cuFD_ga_dx_weights_vector_part(int* tree, double *all_coordinates, double *shapes,int matrix_stride, double* b)
{
//Getting all nearest neighbours in the shared memory
  __shared__ double coordinates[MAX_NN][2];
  int index = tree[blockIdx.x*blockDim.x+threadIdx.x];
  coordinates[threadIdx.x][0] = all_coordinates[index*2] - all_coordinates[blockIdx.x*2];
  coordinates[threadIdx.x][1] = all_coordinates[index*2+1] - all_coordinates[blockIdx.x*2+1];
  __syncthreads();

  int offsetb = blockIdx.x*matrix_stride;
  double x = coordinates[threadIdx.x][0];
  double y = coordinates[threadIdx.x][1];
  b[offsetb+threadIdx.x] = ga_dx(-x,-y,shapes[blockIdx.x]);

  //ugly, pot probably not too harmfull, actually I checked and their is virtually no runtime difference
  if(threadIdx.x == 0)
    {
      for(int i = blockDim.x+2; i < matrix_stride; i++)
	{
	  b[offsetb+i] = 0;
	}
      b[offsetb+blockDim.x] = 0.;
      if((matrix_stride - blockDim.x) > 1)
	{
	  b[offsetb+blockDim.x+1] = 1.;
	}
    }
}

__global__ void cuFD_differentiate_product_pow2(double *f, double *w, double *d)
{
  int index = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double product[MAX_NN];

  product[threadIdx.x] = f[index]*w[index];
  __syncthreads();

  index = blockDim.x /2;
  while(index != 0)
    {
      if(threadIdx.x < index)
	{
	  product[threadIdx.x] += product[threadIdx.x +index];
	}
      __syncthreads();
      index /= 2;
    }
  if(threadIdx.x == 0)
    {
      d[blockIdx.x] = product[0];
    }
}

__global__ void cuFD_differentiate_product(double *f, double *w, double *d)
{
  int index = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double product[MAX_NN];

  product[threadIdx.x] = f[index]*w[index];
  __syncthreads();

  if(threadIdx.x == 0)
    {
      double sum = 0.;
      for(index = 0; index < blockDim.x; index++)
	{
	  sum += product[index];
	}      
      d[blockIdx.x] = sum; 
    }
}



__global__ void cuFD_optimise_const_part(int* tree, double *all_coordinates, int matrix_stride, int pdeg, double* A, double *b, int derivative_order)
{

  int offsetb = blockIdx.x*matrix_stride;
  int offsetA = offsetb*matrix_stride;

  //Getting all nearest neighbours in the shared memory
  __shared__ double coordinates[MAX_NN][2];
  int index = tree[blockIdx.x*blockDim.x+threadIdx.x];
  coordinates[threadIdx.x][0] = all_coordinates[index*2] - all_coordinates[blockIdx.x*2];
  coordinates[threadIdx.x][1] = all_coordinates[index*2+1] - all_coordinates[blockIdx.x*2+1];
  __syncthreads();

  row_vector_from_polynomial(coordinates[threadIdx.x][0],coordinates[threadIdx.x][1],matrix_stride, pdeg,&A[offsetA+threadIdx.x*matrix_stride+blockDim.x],&A[offsetA+blockDim.x*matrix_stride+threadIdx.x]);
  b[offsetb+threadIdx.x] = 0;
 
  //ugly, pot probably not too harmfull, actually I checked and their is virtually no runtime difference
  if(threadIdx.x == 0)
    {
      switch(derivative_order)
	{
	case 1:
	  {
	    for(int i = blockDim.x+2; i < matrix_stride; i++)
	      {
		b[offsetb+i] = 0;
	      }
	    b[offsetb+blockDim.x] = 0.;
	    if((matrix_stride - blockDim.x) > 1)
	      {
		b[offsetb+blockDim.x+1] = 1.;
	      }
	  }
	case 2:
	  {
	    for(int i = blockDim.x+3; i < matrix_stride; i++)
	      {
		b[offsetb+i] = 0;
	      }
	    b[offsetb+blockDim.x] = 0.;
	    if((matrix_stride - blockDim.x) > 2)
	      {
		b[offsetb+blockDim.x+1] = 0.;
		b[offsetb+blockDim.x+2] = 1.;
	      }
	  }
	case 3:
	  {
	    for(int i = blockDim.x+4; i < matrix_stride; i++)
	      {
		b[offsetb+i] = 0;
	      }
	    b[offsetb+blockDim.x] = 0.;
	    if((matrix_stride - blockDim.x) > 2)
	      {
		b[offsetb+blockDim.x+1] = 0.;
		b[offsetb+blockDim.x+2] = 0.;
		if((matrix_stride - blockDim.x) > 3)
		  {
		    b[offsetb+blockDim.x+3] = 2.;
		  }
	      }

	  }
	case 4:
	  {
	    for(int i = blockDim.x+6; i < matrix_stride; i++)
	      {
		b[offsetb+i] = 0;
	      }
	    b[offsetb+blockDim.x] = 0.;
	    if((matrix_stride - blockDim.x) > 2)
	      {
		b[offsetb+blockDim.x+1] = 0.;
		b[offsetb+blockDim.x+2] = 0.;
		if((matrix_stride - blockDim.x) > 5)
		  {
		    b[offsetb+blockDim.x+3] = 0.;
		    b[offsetb+blockDim.x+4] = 0.;
		    b[offsetb+blockDim.x+5] = 2.;
		  }
	      }
	  }
	case 5:
	  {
	    for(int i = blockDim.x+5; i < matrix_stride; i++)
	      {
		b[offsetb+i] = 0;
	      }
	    b[offsetb+blockDim.x] = 0.;
	    if((matrix_stride - blockDim.x) > 2)
	      {
		b[offsetb+blockDim.x+1] = 0.;
		b[offsetb+blockDim.x+2] = 0.;
		if((matrix_stride - blockDim.x) > 4)
		  {
		    b[offsetb+blockDim.x+3] = 0.;
		    b[offsetb+blockDim.x+4] = 1.;
		  }
	      }
	  }
	case 6:
	  {
	    for(int i = blockDim.x+6; i < matrix_stride; i++)
	      {
		b[offsetb+i] = 0;
	      }
	    b[offsetb+blockDim.x] = 0.;
	    if((matrix_stride - blockDim.x) > 2)
	      {
		b[offsetb+blockDim.x+1] = 0.;
		b[offsetb+blockDim.x+2] = 0.;
		if((matrix_stride - blockDim.x) > 5)
		  {
		    b[offsetb+blockDim.x+3] = 2.;
		    b[offsetb+blockDim.x+4] = 0.;
		    b[offsetb+blockDim.x+5] = 2.;
		  }
	      }
	  }
	case 7:
	  {
	    for(int i = blockDim.x+6; i < matrix_stride; i++)
	      {
		b[offsetb+i] = 0;
	      }
	    b[offsetb+blockDim.x] = 0.;
	    if((matrix_stride - blockDim.x) > 2)
	      {
		b[offsetb+blockDim.x+1] = 0.;
		b[offsetb+blockDim.x+2] = 0.;
		if((matrix_stride - blockDim.x) > 5)
		  {
		    b[offsetb+blockDim.x+3] = 2.;
		    b[offsetb+blockDim.x+4] = 0.;
		    b[offsetb+blockDim.x+5] = -2.;
		  }
	      }
	  }
	}
    }
}

__global__ void cuFD_optimise_const_part(int* tree, double *all_coordinates, int matrix_stride, int pdeg, double* A, double *b, int derivative_order, double factor)
{

  int offsetb = blockIdx.x*matrix_stride;
  int offsetA = offsetb*matrix_stride;

  //Getting all nearest neighbours in the shared memory
  __shared__ double coordinates[MAX_NN][2];
  int index = tree[blockIdx.x*blockDim.x+threadIdx.x];
  coordinates[threadIdx.x][0] = all_coordinates[index*2] - all_coordinates[blockIdx.x*2];
  coordinates[threadIdx.x][1] = all_coordinates[index*2+1] - all_coordinates[blockIdx.x*2+1];
  __syncthreads();

  row_vector_from_polynomial(coordinates[threadIdx.x][0],coordinates[threadIdx.x][1],matrix_stride, pdeg,&A[offsetA+threadIdx.x*matrix_stride+blockDim.x],&A[offsetA+blockDim.x*matrix_stride+threadIdx.x]);
  b[offsetb+threadIdx.x] = 0;
 
  //ugly, pot probably not too harmfull, actually I checked and their is virtually no runtime difference
  if(threadIdx.x == 0)
    {
      switch(derivative_order)
	{
	case 1:
	  {
	    for(int i = blockDim.x+2; i < matrix_stride; i++)
	      {
		b[offsetb+i] = 0;
	      }
	    b[offsetb+blockDim.x] = 0.;
	    if((matrix_stride - blockDim.x) > 1)
	      {
		b[offsetb+blockDim.x+1] = factor;
	      }
	  }
	case 2:
	  {
	    for(int i = blockDim.x+3; i < matrix_stride; i++)
	      {
		b[offsetb+i] = 0;
	      }
	    b[offsetb+blockDim.x] = 0.;
	    if((matrix_stride - blockDim.x) > 2)
	      {
		b[offsetb+blockDim.x+1] = 0.;
		b[offsetb+blockDim.x+2] = factor;
	      }
	  }
	case 3:
	  {
	    for(int i = blockDim.x+4; i < matrix_stride; i++)
	      {
		b[offsetb+i] = 0;
	      }
	    b[offsetb+blockDim.x] = 0.;
	    if((matrix_stride - blockDim.x) > 2)
	      {
		b[offsetb+blockDim.x+1] = 0.;
		b[offsetb+blockDim.x+2] = 0.;
		if((matrix_stride - blockDim.x) > 3)
		  {
		    b[offsetb+blockDim.x+3] = 2.*factor;
		  }
	      }

	  }
	case 4:
	  {
	    for(int i = blockDim.x+6; i < matrix_stride; i++)
	      {
		b[offsetb+i] = 0;
	      }
	    b[offsetb+blockDim.x] = 0.;
	    if((matrix_stride - blockDim.x) > 2)
	      {
		b[offsetb+blockDim.x+1] = 0.;
		b[offsetb+blockDim.x+2] = 0.;
		if((matrix_stride - blockDim.x) > 5)
		  {
		    b[offsetb+blockDim.x+3] = 0.;
		    b[offsetb+blockDim.x+4] = 0.;
		    b[offsetb+blockDim.x+5] = 2.*factor;
		  }
	      }
	  }
	case 5:
	  {
	    for(int i = blockDim.x+5; i < matrix_stride; i++)
	      {
		b[offsetb+i] = 0;
	      }
	    b[offsetb+blockDim.x] = 0.;
	    if((matrix_stride - blockDim.x) > 2)
	      {
		b[offsetb+blockDim.x+1] = 0.;
		b[offsetb+blockDim.x+2] = 0.;
		if((matrix_stride - blockDim.x) > 4)
		  {
		    b[offsetb+blockDim.x+3] = 0.;
		    b[offsetb+blockDim.x+4] = factor;
		  }
	      }
	  }
	case 6:
	  {
	    for(int i = blockDim.x+6; i < matrix_stride; i++)
	      {
		b[offsetb+i] = 0;
	      }
	    b[offsetb+blockDim.x] = 0.;
	    if((matrix_stride - blockDim.x) > 2)
	      {
		b[offsetb+blockDim.x+1] = 0.;
		b[offsetb+blockDim.x+2] = 0.;
		if((matrix_stride - blockDim.x) > 5)
		  {
		    b[offsetb+blockDim.x+3] = 2.*factor;
		    b[offsetb+blockDim.x+4] = 0.;
		    b[offsetb+blockDim.x+5] = 2.*factor;
		  }
	      }
	  }
	case 7:
	  {
	    for(int i = blockDim.x+6; i < matrix_stride; i++)
	      {
		b[offsetb+i] = 0;
	      }
	    b[offsetb+blockDim.x] = 0.;
	    if((matrix_stride - blockDim.x) > 2)
	      {
		b[offsetb+blockDim.x+1] = 0.;
		b[offsetb+blockDim.x+2] = 0.;
		if((matrix_stride - blockDim.x) > 5)
		  {
		    b[offsetb+blockDim.x+3] = 2.*factor;
		    b[offsetb+blockDim.x+4] = 0.;
		    b[offsetb+blockDim.x+5] = -2.*factor;
		  }
	      }
	  }
	}
    }
}

__global__ void cuFD_optimise_func_eval(int *tree, double *func, double *nn_func)
{
  int index = blockIdx.x*blockDim.x+threadIdx.x;
  nn_func[index] = func[tree[index]];
}

__global__ void cuFD_optimise_calculate_and_compare(double *f, double *w, double *truth, double *err, int matrix_stride)
{

  int offset_w = blockIdx.x*matrix_stride;
  int offset_f = blockIdx.x*blockDim.x;
  __shared__ double product[MAX_NN];

  product[threadIdx.x] = f[offset_f+threadIdx.x]*w[offset_w+threadIdx.x];
  __syncthreads();
  //This is certainly not ideal, in the following we also provide a routine which iteratively sums this up. However, the number of threads in the block must be a power 2 in this case. 
  if(threadIdx.x == 0)
    {
      double sum = 0.;
      for(int i = 0; i < blockDim.x; i++)
	{
	  sum += product[i];
	}
      double correct = truth[blockIdx.x]; 
      err[blockIdx.x] = abs(sum - correct); 
    }
}

__global__ void cuFD_optimise_calculate_and_compare_pow2(double *f, double *w, double *truth, double *err, int matrix_stride)
{

  int offset_w = blockIdx.x*matrix_stride;
  int offset_f = blockIdx.x*blockDim.x;
  __shared__ double product[MAX_NN];

  product[threadIdx.x] = f[offset_f+threadIdx.x]*w[offset_w+threadIdx.x];
  __syncthreads();

  int i = blockDim.x /2;
  while(i != 0)
    {
      if(threadIdx.x < i)
	{
	  product[threadIdx.x] += product[threadIdx.x +i];
	}
      __syncthreads();
      i /= 2;
    }

  if(threadIdx.x == 0)
    {
      double correct = truth[blockIdx.x]; 
      err[blockIdx.x] = abs(product[0] - correct); 
    }
}




