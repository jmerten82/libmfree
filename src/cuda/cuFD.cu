/*** /cuda/cuFD.cu
     A collection of host drivers which 
     perform RBF FD operations.

Julian Merten
INAF OA Bologna
Jul 2018
julian.merten@oabo.inaf.it
http://www.julianmerten.net
***/

#include <mfree/cuda/cuFD.h>

void cuFD_differentiate_set(vector<double> function, cuda_manager *cuman)
{
  int num_nodes = cuman->n("nodes");
  int nn = cuman->n("nn");

  //Very simple error checks
  if(function.size() < num_nodes)
    {
      throw invalid_argument("cuFD: Given function too short for managed grid.");
    }

  //Setting device to first one
  checkCudaErrors(hipSetDevice(cuman->device_query(0).second));

  //Getting f on device. 
  double *d_f;
  checkCudaErrors(hipMalloc((void **)&d_f,sizeof(double)*num_nodes));
  checkCudaErrors(hipMemcpy(d_f,&function[0],num_nodes*sizeof(double),hipMemcpyHostToDevice));

  //Running kernel to set function
  cuFD_optimise_func_eval<<<num_nodes,nn>>>(cuman->index_map_pointer(),d_f,cuman->FD_device_pointer("function"));

  //Distributing function to all devices
  cuman->distribute_FD("function");

  //Free allocated memory
  checkCudaErrors(hipFree(d_f));

}

vector<double> cuFD_differentiate_and_return(int derivative_selection, cuda_manager *cuman)
{
  string selection;
  switch(derivative_selection)
    {
    case 1: selection = "dx";
    case 2: selection = "dy";
    case 3: selection = "dxx";
    case 4: selection = "dxy";
    case 5: selection = "dxy";
    case 6: selection = "laplace";
    case 7: selection = "neg_laplace";
    }

  //numbers
  int num_nodes = cuman->n("nodes");
  int nn = cuman->n("nn");
  
  //Setting device to first one
  checkCudaErrors(hipSetDevice(cuman->device_query(0).second));

  if(nn == 2 || nn == 4 || nn == 8 || nn == 16 || nn == 32 || nn == 64 || nn == 128)
    {
      cuFD_differentiate_product_pow2<<<num_nodes,nn>>>(cuman->FD_device_pointer("function"),cuman->FD_device_pointer(selection),cuman->FD_device_pointer("derivative"));
    }
  else
    {
      cuFD_differentiate_product_pow2<<<num_nodes,nn>>>(cuman->FD_device_pointer("function"),cuman->FD_device_pointer(selection),cuman->FD_device_pointer("derivative"));
    }

  //Sending result around to all devices
  cuman->distribute_FD("derivative");

  //Getting derivative off device
  vector<double> result(num_nodes,0.);
  checkCudaErrors(hipMemcpy(&result[0],cuman->FD_device_pointer("derivative"),num_nodes*sizeof(double),hipMemcpyDeviceToHost));

  return result;

}


void cuFD_differentiate(int derivative_selection, cuda_manager *cuman)
{
  string selection;
  switch(derivative_selection)
    {
    case 1: selection = "dx";
    case 2: selection = "dy";
    case 3: selection = "dxx";
    case 4: selection = "dxy";
    case 5: selection = "dxy";
    case 6: selection = "laplace";
    case 7: selection = "neg_laplace";
    }

  //numbers
  int num_nodes = cuman->n("nodes");
  int nn = cuman->n("nn");

  //Setting device to first one
  checkCudaErrors(hipSetDevice(cuman->device_query(0).second));

  //Calculating derivatives in the fastest way depending on nearest neighbours
  if(nn == 2 || nn == 4 || nn == 8 || nn == 16 || nn == 32 || nn == 64 || nn == 128)
    {
      cuFD_differentiate_product_pow2<<<num_nodes,nn>>>(cuman->FD_device_pointer("function"),cuman->FD_device_pointer(selection),cuman->FD_device_pointer("derivative"));
    }
  else
    {
      cuFD_differentiate_product<<<num_nodes,nn>>>(cuman->FD_device_pointer("function"),cuman->FD_device_pointer(selection),cuman->FD_device_pointer("derivative"));
    }
  cuman->distribute_FD("derivative");
}


vector<vector<double> > cuFD_differentiate(cuda_manager *cuman)
{

}
